#include "hip/hip_runtime.h"
#include "Image.h"
#include "PPM.h"
#include <iostream>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <math.h>
using namespace std;
using namespace std:: chrono;


#define TILE_WIDTH 16
#define maskCols 5
#define maskRows 5
#define w (TILE_WIDTH + maskCols -1)


__global__ void tilingKernelProcessing(float * InputImageData, const float *__restrict__ kernel,
		float* outputImageData, int channels, int width, int height){

	__shared__ float N_ds[w][w];  //block of image in shared memory


	// allocation in shared memory of image blocks
	int maskRadius = maskRows/2;
 	for (int k = 0; k <channels; k++) {
 		int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
 		int destY = dest/w;     //row of shared memory
 		int destX = dest%w;		//col of shared memory
 		int srcY = blockIdx.y *TILE_WIDTH + destY - maskRadius; // index to fetch data from input image
 		int srcX = blockIdx.x *TILE_WIDTH + destX - maskRadius; // index to fetch data from input image
 		int src = (srcY *width +srcX) * channels + k;   // index of input image
 		if(srcY>= 0 && srcY < height && srcX>=0 && srcX < width)
 			N_ds[destY][destX] = InputImageData[src];  // copy element of image in shared memory
 		else
 			N_ds[destY][destX] = 0;



 		dest = threadIdx.y * TILE_WIDTH+ threadIdx.x + TILE_WIDTH * TILE_WIDTH;
 		destY = dest/w;
		destX = dest%w;
		srcY = blockIdx.y *TILE_WIDTH + destY - maskRadius;
		srcX = blockIdx.x *TILE_WIDTH + destX - maskRadius;
		src = (srcY *width +srcX) * channels + k;
		if(destY < w){
			if(srcY>= 0 && srcY < height && srcX>=0 && srcX < width)
				N_ds[destY][destX] = InputImageData[src];
			else
				N_ds[destY][destX] = 0;
		}

 		__syncthreads();


 		//compute kernel convolution
 		float accum = 0;
 		int y, x;
 		for (y= 0; y < maskCols; y++)
 			for(x = 0; x<maskRows; x++)
 				accum += N_ds[threadIdx.y + y][threadIdx.x + x] *kernel[y * maskCols + x];

 		y = blockIdx.y * TILE_WIDTH + threadIdx.y;
 		x = blockIdx.x * TILE_WIDTH + threadIdx.x;
 		if(y < height && x < width)
 			outputImageData[(y * width + x) * channels + k] = accum;
 		__syncthreads();


 	}

}




int main(){

	int imageChannels;
	int imageHeight;
	int imageWidth;
	Image_t* inputImage;
	Image_t* outputImage;
	float* hostInputImageData;
	float* hostOutputImageData;
	float* deviceInputImageData;
	float* deviceOutputImageData;
	float* deviceMaskData;
	float hostMaskData[maskRows * maskCols]={
			0.04, 0.04, 0.04, 0.04, 0.04,
			0.04, 0.04, 0.04, 0.04, 0.04,
			0.04, 0.04, 0.04, 0.04, 0.04,
			0.04, 0.04, 0.04, 0.04, 0.04,
			0.04, 0.04, 0.04, 0.04, 0.04


	};


	inputImage = PPM_import("./img/computer_programming.ppm");

	imageWidth = Image_getWidth(inputImage);
	imageHeight = Image_getHeight(inputImage);
	imageChannels = Image_getChannels(inputImage);

	outputImage = Image_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = Image_getData(inputImage);
	hostOutputImageData = Image_getData(outputImage);

	hipDeviceReset();
	hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight *
			imageChannels * sizeof(float));
	hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight *
			imageChannels * sizeof(float));
	hipMalloc((void **) &deviceMaskData, maskRows * maskCols
			* sizeof(float));
	hipMemcpy(deviceInputImageData, hostInputImageData,
			imageWidth * imageHeight * imageChannels * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData, hostMaskData,
				maskRows * maskCols * sizeof(float),
				hipMemcpyHostToDevice);


	dim3 dimGrid(ceil((float) imageWidth/TILE_WIDTH),
			ceil((float) imageHeight/TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);


	cout <<"SHARED MEMORY KERNEL CONVOLUTION" << endl;
	cout << "image dimensions: "<< imageWidth << "x" << imageHeight << endl;
	cout << "start parallelizing" << endl;
	cout << "elapsed in time: ";
	high_resolution_clock::time_point start= high_resolution_clock::now();

	tilingKernelProcessing<<<dimGrid,dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,
	imageChannels, imageWidth, imageHeight);

	high_resolution_clock::time_point end= high_resolution_clock::now();
	chrono::duration<double>  duration = end - start;
	cout << duration.count()*1000 << endl;
	cout << "----------------------------------" << endl;

	hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight *
			imageChannels * sizeof(float), hipMemcpyDeviceToHost);

	PPM_export("./output/result.ppm", outputImage);

	hipMemset(deviceInputImageData,0,imageWidth * imageHeight *
				imageChannels * sizeof(float));
	hipMemset(deviceOutputImageData,0,imageWidth * imageHeight *
					imageChannels * sizeof(float));
	hipMemset(deviceMaskData,0,maskRows * maskCols
				* sizeof(float));
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceMaskData);

	Image_delete(outputImage);
	Image_delete(inputImage);


}
